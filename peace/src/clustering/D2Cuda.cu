#include "hip/hip_runtime.h"
#ifndef D2_CUDA_CU
#define D2_CUDA_CU

//--------------------------------------------------------------------
//
// This file is part of PEACE.
// 
// PEACE is free software: you can redistribute it and/or modify it
// under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
// 
// PEACE is distributed in the hope that it will be useful, but
// WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
// General Public License for more details.
// 
// You should have received a copy of the GNU General Public License
// along with PEACE.  If not, see <http://www.gnu.org/licenses/>.
// 
// Miami University makes no representations or warranties about the
// suitability of the software, either express or implied, including
// but not limited to the implied warranties of merchantability,
// fitness for a particular purpose, or non-infringement.  Miami
// University shall not be liable for any damages suffered by licensee
// as a result of using, result of using, modifying or distributing
// this software or its derivatives.
//
// By using or copying this Software, Licensee agrees to abide by the
// intellectual property laws, and all other applicable laws of the
// U.S., and the terms of GNU General Public License (version 3).
//
// Authors: Tuan Le                   letm@miamioh.edu
//          Dhananjai M. Rao          raodm@miamioh.edu
//---------------------------------------------------------------------

#include "D2Cuda.h"
#include "ESTCodec.h"
#include "ArgParser.h"
#include "HeuristicChain.h"
#include <algorithm>

D2Cuda::D2Cuda() : FWAnalyzer("D2Cuda") {
    alignmentMetric  = 0;
    threshold        = 0;
    frameShift       = 1;
    // Fix frame and word size
    frameSize        = 100;
    wordSize         = 6;
    bitMask          = (1 << (wordSize * 2)) - 1; // 4095
    numWordsInWindow = frameSize - wordSize + 1;
}

D2Cuda::~D2Cuda() {

}

void D2Cuda::addCommandLineArguments(ArgParser& argParser) {
    // Let base class add common parameters.
    FWAnalyzer::addCommandLineArguments(argParser);
    // Now add our custom parameters.
    const ArgParser::ArgRecord ArgsList[] = {
        {"--frameShift", "Frame Shift (default=1)",
         &frameShift, ArgParser::INTEGER},
        {"--d2Threshold", "Threshold score to break out of D2Cuda (default=0)",
         &threshold, ArgParser::INTEGER},
        {"", "", NULL, ArgParser::INVALID}
    };
    argParser.addValidArguments(ArgsList);
}

bool D2Cuda::initialize() {
    // Let the base class initialize any additional heuristics
    if (!FWAnalyzer::initialize()) {
        // Error occured when initializing.  This is no good.
        return false;
    }
    // Ensure frameshift is valid.
    if (frameShift < 1) {
        std::cerr << getName()
                  << ": Frame shift must be >= 1"
                  << "(use --frameShift option)\n";
        return false;
    }
    // Everything went on well.
    return true;
}

/** A helper method to check for GPU errors
    Sample usage:
        gpuErrCheck(hipMalloc(...));
        gpuErrCheck(hipPeekAtLastError());
*/
#define gpuErrCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
    if (code != hipSuccess) {
        fprintf(stderr,"GPUassert: %s at %s: %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

/** Method to encode a character to its numerical representation.
    Since this method is only used in buildWordTable, it's annotated with __device__,
    meaning it can only be called on GPU.
*/
__device__ void encode(char c, int &v) {
    if (c == 'A' || c == 'a') v = 0;
    else if (c == 'C' || c == 'c') v = 1;
    else if (c == 'G' || c == 'g') v = 2;
    else if (c == 'T' || c == 't') v = 3;
    else v = -1;
}

/** Method to fill a word table with a sequence's words' numerical representations.
    Since this method is called from host code, it's annotated with __global__.
*/
__global__ void buildWordTable(int *wordTable, char *seq, int numWords, int wordSize, int bitMask) {
    int start = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = start; i < numWords; i += stride) {
        int hash = 0, value;
        bool ignoreHash = false;
        for (int j = i; j < i + wordSize; j++) {
            encode(seq[j], value);
            if (value == -1) {
                ignoreHash = true;
                break;
            }
            hash = ((hash << 2) | value) & bitMask;
        }
        if (!ignoreHash) {
            wordTable[i] = hash;
        }
    }
}

int D2Cuda::setReferenceEST(const EST* est) {
    ASSERT ( est != NULL );
    // Call corresponding method in heuristic chain
    if (chain != NULL) {
        chain->setReferenceEST(est);
    }
    refEST = est;
    // Init ref-est word table
    const std::string s1Str = est->getSequenceString();
    // Calculate length and number of words in sequence
    int s1Length = s1Str.size();
    int numWordsInS1 = s1Length - wordSize + 1;
    numWindowsInS1 = s1Length - frameSize + 1;
    // Allocate the word table on CUDA
    hipMalloc(&s1WordTable, sizeof(int) * numWordsInS1);
    // Allocate the sequence on CUDA so we can read it inside kernel
    char *s1;
    hipMalloc(&s1, sizeof(char) * s1Length);
    hipMemcpy(s1, s1Str.c_str(), sizeof(char) * s1Length, hipMemcpyHostToDevice);
    // Calculate number of blocks, rounded up
    int numBlocks = (numWordsInS1 + numThreads - 1) / numThreads;
    // Call the actual buildWordTable kernel
    buildWordTable<<<numBlocks, numThreads>>>(s1WordTable, s1, numWordsInS1, wordSize, bitMask);
    // Wait for CUDA to finish and free the sequence memory
    hipDeviceSynchronize();
    hipFree(s1);

    return 0;
}

float D2Cuda::getMetric(const EST* otherEST) {
    ASSERT ( otherEST != NULL );
    VALIDATE({
        if (otherEST->getID() == refEST->getID()) {
            return 0; // distance to self will be 0
        }
    });
    // OK. Run the actual d2 algorithm
    return (float) runD2(otherEST);
}

/** Method to calculate min D2 score. For each thread running this method, it picks
    a pair of windows from two sequences and calculate D2 score for this pair.
    A delta array is used to keep track of words count differences between two windows.
    Its size is fixed to 4096 since a word largest possible numerical representation is 4095.
    Since this method is called from host code, it's annotated with __global__.
*/
__global__ void getScore(int* s1WordTable, int* s2WordTable, int* minScore, bool* done, int numWindowsInS1, int numWindowsInS2, int numWordsInWindow, int threshold) {
    int start = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    int delta[4096];
    // Array values seem to be already initialized with 0
    // Writing to array in GPU is very slow
    // for (int i = 0; i < 4096; i++) {
    //     delta[i] = 0;
    // }
    for (int i = start; i < numWindowsInS1 * numWindowsInS2; i += stride) {
        if (*done) {
            return;
        }
        int s1WindowIndex = i % numWindowsInS1, s2WindowIndex = i / numWindowsInS1, j;
        for (j = s1WindowIndex; j < s1WindowIndex + numWordsInWindow; j++) {
            delta[s1WordTable[j]] += 1;
        }
        for (j = s2WindowIndex; j < s2WindowIndex + numWordsInWindow; j++) {
            delta[s2WordTable[j]] -= 1;
        }
        int score = 0;
        for (j = 0; j < 4096; j++) {
            if (delta[j] != 0) {
                score += delta[j] * delta[j];
                delta[j] = 0;
            }
        }
        atomicMin(minScore, score);
        if (score <= threshold) {
            *done = true;
        }
    }
}

float D2Cuda::runD2(const EST *otherEST) {
    const std::string s2Str = otherEST->getSequenceString();
    // Calculate length and number of words in sequence
    int s2Length = s2Str.size();
    int numWordsInS2 = s2Length - wordSize + 1;
    numWindowsInS2 = s2Length - frameSize + 1;
    // Allocate the word table on CUDA
    hipMalloc(&s2WordTable, sizeof(int) * numWordsInS2);
    // Allocate the sequence on CUDA so we can read it inside kernel
    char *s2;
    hipMalloc(&s2, sizeof(char) * s2Length);
    hipMemcpy(s2, s2Str.c_str(), sizeof(char) * s2Length, hipMemcpyHostToDevice);
    // Calculate number of blocks, rounded up
    int numBlocks = (numWordsInS2 + numThreads - 1) / numThreads;
    // Call the actual kernel
    buildWordTable<<<numBlocks, numThreads>>>(s2WordTable, s2, numWordsInS2, wordSize, bitMask);
    // Wait for CUDA to finish and free the sequence memory
    hipDeviceSynchronize();
    hipFree(s2);

    bool* done;
    int* minScore;
    hipMalloc(&done, sizeof(bool));
    hipMallocManaged(&minScore, sizeof(int));
    *minScore = INT_MAX;

    numBlocks = (numWindowsInS1 * numWindowsInS2 + numThreads - 1) / numThreads;
    getScore<<<numBlocks, numThreads>>>(s1WordTable, s2WordTable, minScore, done, numWindowsInS1, numWindowsInS2, numWordsInWindow, threshold);

    hipDeviceSynchronize();

    int score = *minScore;

    hipFree(s1WordTable);
    hipFree(s2WordTable);
    hipFree(minScore);
    hipFree(done);

    return (float) score;
}

bool D2Cuda::getAlignmentData(int &alignmentData) {
    // Simply copy the alignment metric that was computed by the last
    // successful call to the analyze() method.
    alignmentData = alignmentMetric;
    // Let the caller know that the alignment data is available.
    return true;
}

#endif
