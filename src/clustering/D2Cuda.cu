#ifndef D2_CUDA_CU
#define D2_CUDA_CU

//--------------------------------------------------------------------
//
// This file is part of PEACE.
// 
// PEACE is free software: you can redistribute it and/or modify it
// under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
// 
// PEACE is distributed in the hope that it will be useful, but
// WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
// General Public License for more details.
// 
// You should have received a copy of the GNU General Public License
// along with PEACE.  If not, see <http://www.gnu.org/licenses/>.
// 
// Miami University makes no representations or warranties about the
// suitability of the software, either express or implied, including
// but not limited to the implied warranties of merchantability,
// fitness for a particular purpose, or non-infringement.  Miami
// University shall not be liable for any damages suffered by licensee
// as a result of using, result of using, modifying or distributing
// this software or its derivatives.
//
// By using or copying this Software, Licensee agrees to abide by the
// intellectual property laws, and all other applicable laws of the
// U.S., and the terms of GNU General Public License (version 3).
//
// Authors: Tuan Le                   letm@miamioh.edu
//          Dhananjai M. Rao          raodm@miamioh.edu
//---------------------------------------------------------------------

#include "D2Cuda.h"
#include "ESTCodec.h"
#include "ArgParser.h"
#include "HeuristicChain.h"
#include <algorithm>

// The bitmak to be used when build hash values.
int D2Cuda::BitMask   = 0;
// Instance variable to store the number of bits to be shifted to
// create hash values. This value is initialized to 2*(wordSize-1)
int D2Cuda::bitShift  = 0;

D2Cuda::D2Cuda() : FWAnalyzer("D2Cuda") {
    delta           = NULL;
    alignmentMetric = 0;
    threshold       = 0;
    frameShift      = 1;
}

D2Cuda::~D2Cuda() {
    if (delta != NULL) {
        delete [] delta;
    }
}

void
D2Cuda::addCommandLineArguments(ArgParser& argParser) {
    // Let base class add common parameters.
    FWAnalyzer::addCommandLineArguments(argParser);
    // Now add our custom parameters.
    const ArgParser::ArgRecord ArgsList[] = {
        {"--frameShift", "Frame Shift (default=1)",
         &frameShift, ArgParser::INTEGER},
        {"--d2Threshold", "Threshold score to break out of D2Cuda (default=0)",
         &threshold, ArgParser::INTEGER},    
        {"", "", NULL, ArgParser::INVALID}
    };
    argParser.addValidArguments(ArgsList);
}

bool
D2Cuda::initialize() {
    // Let the base class initialize any additional heuristics
    if (!FWAnalyzer::initialize()) {
        // Error occured when initializing.  This is no good.
        return false;
    }
    // Ensure frameshift is valid.
    if (frameShift < 1) {
        std::cerr << getName()
                  << ": Frame shift must be >= 1"
                  << "(use --frameShift option)\n";
        return false;
    }
    
    // Setup the frequency delta table
    const int MapSize = (1 << (wordSize * 2));
    delta = new int[MapSize];    
    // Compute bit mask that will retain only the bits corresponding
    // to a given word size.  Each entry in a word takes up 2 bits and
    // that is why the following formula involves a 2.
    BitMask = (1 << (wordSize * 2)) - 1;
    // Compute the number of bits to shift when building hashes
    bitShift = 2 * (wordSize - 1);
    // Set the number of words in a window.
    numWordsInWindow = frameSize - wordSize + 1;    
    // Everything went on well.
    return true;
}

int
D2Cuda::setReferenceEST(const EST* est) {
    ASSERT ( est != NULL );
    // Call corresponding method in heuristic chain
    if (chain != NULL) {
        chain->setReferenceEST(est);
    }
    refEST = est;
    // init ref-est word table
    const char* s1   = est->getSequence();

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    std::cout << prop.name << std::endl;

    ASSERT("Implement me" == NULL);

    return 1; // For now, return non-zero to signify error
}

float
D2Cuda::getMetric(const EST* otherEST) {
    ASSERT ( otherEST != NULL );
    VALIDATE({
        if (otherEST->getID() == refEST->getID()) {
            return 0; // distance to self will be 0
        }
    });
    
    // OK. Run the actual d2 algorithm
    return (float) runD2(otherEST);
}

float
D2Cuda::runD2(const EST* estS2) {
    return 0;  // Currently unimplemented.
}

bool
D2Cuda::getAlignmentData(int &alignmentData) {
    // Simply copy the alignment metric that was computed by the last
    // successful call to the analyze() method.
    alignmentData = alignmentMetric;
    // Let the caller know that the alignment data is available.
    return true;
}

#endif
